#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;

// Definir la estructura de un punto en el espacio
typedef struct {
    double x;
    double y;
} Point;

// Función para calcular la distancia euclidiana entre dos puntos
__device__ double euclidean_distance(Point p1, Point p2) {
    double dx = p1.x - p2.x;
    double dy = p1.y - p2.y;
    return sqrt(dx * dx + dy * dy);
}

__global__ void assign_points_kernel(Point* points, int num_points, Point* centroids, int num_centroids, int* assignments) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_points) {
        double min_distance = INFINITY;
        int centroid_index = 0;
        
        for (int j = 0; j < num_centroids; j++) {
            double distance = euclidean_distance(points[i], centroids[j]);
            
            if (distance < min_distance) {
                min_distance = distance;
                centroid_index = j;
            }
        }
        
        assignments[i] = centroid_index;
    }
}

void assign_points(Point* points, int num_points, Point* centroids, int num_centroids, int* assignments, int iteration) {
    if (iteration < 30) {
        printf("asignando puntos...\n");
    }

    Point* d_points;
    Point* d_centroids;
    int* d_assignments;
    
    // Alojar memoria en el dispositivo
    hipMalloc((void**)&d_points, num_points * sizeof(Point));
    hipMalloc((void**)&d_centroids, num_centroids * sizeof(Point));
    hipMalloc((void**)&d_assignments, num_points * sizeof(int));
    
    // Copiar datos desde el host al dispositivo
    hipMemcpy(d_points, points, num_points * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, num_centroids * sizeof(Point), hipMemcpyHostToDevice);
    
    // Lanzar el kernel de CUDA
    int block_size = 256;
    int num_blocks = (num_points + block_size - 1) / block_size;
    assign_points_kernel<<<num_blocks, block_size>>>(d_points, num_points, d_centroids, num_centroids, d_assignments);
    
    // Copiar los resultados desde el dispositivo al host
    hipMemcpy(assignments, d_assignments, num_points * sizeof(int), hipMemcpyDeviceToHost);
    
    // Liberar memoria en el dispositivo
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_assignments);
}


// Función para recalcular las posiciones de los centroides
void update_centroids(Point* points, int num_points, Point* centroids, int num_centroids, int* assignments, int iteration) {
    int* counts = (int*)calloc(num_centroids, sizeof(int));
    double* sum_x = (double*)calloc(num_centroids, sizeof(double));
    double* sum_y = (double*)calloc(num_centroids, sizeof(double));
    if(iteration < 30){
        printf("Actualizando centroides...\n\n");
    }
 
    for (int i = 0; i < num_points; i++) {
        int centroid_index = assignments[i];
        counts[centroid_index]++;
        sum_x[centroid_index] += points[i].x;
        sum_y[centroid_index] += points[i].y;
    }

    for (int i = 0; i < num_centroids; i++) {
        
        if (counts[i] > 0) {
            centroids[i].x = sum_x[i] / counts[i];
            centroids[i].y = sum_y[i] / counts[i];
        }
    }
    free(counts);
    free(sum_x);
    free(sum_y);
}

// Función para imprimir los resultados del algoritmo
void print_results(Point* points, int num_points, Point* centroids, int num_centroids, int* assignments) {
    remove("datos.txt");
    remove("centroide.txt");
    FILE* file = fopen("datos.txt", "a");
    if (file == NULL) {
        printf("No se pudo crear el archivo.\n");
        return;
    }
    FILE* fileCentroide = fopen("centroide.txt", "a");
    if (fileCentroide == NULL) {
        printf("No se pudo crear el archivo.\n");
        return;
    }
    for (int i = 0; i < num_points; i++) {
       fprintf(file, "%.2f;%.2f;%d\n",points[i].x, points[i].y, assignments[i]);
    }
    
    printf("Guardando resultados finales.......\n");
    for (int i = 0; i < num_centroids; i++) {
        fprintf(fileCentroide, "C%d;%.2f;%.2f\n", i + 1, centroids[i].x, centroids[i].y);
    }
    fclose(file);
    fclose(fileCentroide);
}

int main(int argc, char **argv) {
    if(argc != 5){
        fprintf(stderr, "run as ./prog nc n it nt seed\nnc = número de centroides\nn = número de elementos\nit = numero de iteraciones\nseed = semilla\n");
        exit(EXIT_SUCCESS);
    }
    
    // Configuración del algoritmo
    int nc = atoi(argv[1]);
    int n = atoi(argv[2]);
    int it = atoi(argv[3]);
    int seed = atoi(argv[4]);

    int num_centroids = nc;         // Número de centroides
    int num_points = n;            // Número de puntos
    int max_iterations = it;       // Número máximo de iteraciones
    double min_value = 0.0;        // Valor mínimo para generar puntos aleatorios
    double max_value = 10.0;       // Valor máximo para generar puntos aleatorios

    // Creación de los puntos aleatorios
    mt19937_64 drng;
    drng.seed(seed);
    uniform_real_distribution<double> dist(min_value, max_value);

    Point* points = (Point*)malloc(num_points * sizeof(Point));
    printf("Generando datos aleatorios... \n\n");
    for (int i = 0; i < num_points; i++) {
        points[i].x = dist(drng);
        points[i].y = dist(drng);
    }
    
    // Creación de los centroides iniciales
    Point* centroids = (Point*)malloc(num_centroids * sizeof(Point));
    printf("Inicializando centroides... \n\n");
    for (int i = 0; i < num_centroids; i++) {
        centroids[i].x = dist(drng);
        centroids[i].y = dist(drng);
    }
    
    // Asignación inicial de puntos a centroides
    int* assignments = (int*)malloc(num_points * sizeof(int));

    // Medir el tiempo de ejecución
    printf("Comienza el cálculo... \n\n");
    clock_t start_time = clock();

    // Bucle principal del algoritmo
    int iteration = 0;
    while (iteration < max_iterations) {
        if(max_iterations < 30){
            printf("Iteracion %i\n", iteration+1);
        }
        assign_points(points, num_points, centroids, num_centroids, assignments, max_iterations);
        update_centroids(points, num_points, centroids, num_centroids, assignments, max_iterations);
        iteration++;
    }
    
    // Calcular el tiempo transcurrido en segundos
    clock_t end_time = clock();
    double execution_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    // Imprimir resultados
    printf("Tiempo de ejecución: %.2f segundos\n", execution_time);
    print_results(points, num_points, centroids, num_centroids, assignments);
    
    // Liberar memoria
    free(points);
    free(centroids);
    free(assignments);
    
    return 0;
}